PIC_LD=ld

ARCHIVE_OBJS=

VCS_ARC0 =_csrc0.so

VCS_OBJS0 =objs/FkrLU_d.o objs/S4z5S_d.o objs/jrpm8_d.o objs/qTaKS_d.o objs/V0ZpU_d.o  \
objs/TSdIn_d.o objs/IktGB_d.o objs/SvjDT_d.o objs/IvMgd_d.o objs/RDfSw_d.o  \
objs/ve49z_d.o objs/epAHI_d.o objs/YpTIH_d.o objs/tMWkg_d.o objs/gfDqW_d.o  \
objs/R959q_d.o objs/x5r7N_d.o objs/cutvU_d.o objs/iG8SZ_d.o objs/GamDj_d.o  \
objs/cW0NN_d.o objs/iuqnp_d.o objs/xMKmz_d.o objs/naJ9I_d.o objs/ExIS6_d.o  \
objs/vqJAs_d.o objs/AejSG_d.o objs/d3WsF_d.o objs/EpCZ7_d.o objs/VNTpu_d.o  \
objs/gU23g_d.o objs/LKIve_d.o objs/qzayT_d.o objs/kvEUG_d.o objs/GUnBF_d.o  \
objs/uSH1z_d.o objs/r8iH3_d.o objs/ENJPm_d.o objs/A55h6_d.o objs/pAFVC_d.o  \
objs/wI2UF_d.o objs/rCyYs_d.o objs/fYyAz_d.o objs/EiDw9_d.o objs/VqeKH_d.o  \
objs/VuCm7_d.o objs/HUt6f_d.o objs/UxugA_d.o objs/ZAEjR_d.o objs/kLJYY_d.o  \
objs/rrgR2_d.o objs/Iy7i5_d.o objs/K5KAI_d.o objs/M4bka_d.o objs/QpjT4_d.o  \
objs/Q0r7E_d.o objs/yWvC4_d.o objs/td6yT_d.o objs/DpEE4_d.o objs/Qpgvp_d.o  \
objs/DW7Nj_d.o objs/F3GL0_d.o objs/i4j0p_d.o objs/fDKmn_d.o objs/RBNGa_d.o  \
objs/hP6N4_d.o objs/WpKcM_d.o objs/bW24P_d.o objs/hyMdT_d.o objs/TsZEw_d.o  \
objs/VBr21_d.o objs/axK6x_d.o objs/M0zze_d.o objs/drbh5_d.o objs/JtWie_d.o  \
objs/QpMM1_d.o objs/QPQL6_d.o objs/IvgI8_d.o objs/WbMrJ_d.o objs/wiTJJ_d.o  \
objs/Jqca4_d.o objs/DWbdD_d.o objs/UbK6T_d.o objs/cVD8U_d.o objs/PuFsy_d.o  \
objs/FGvwa_d.o objs/Zfv0P_d.o objs/U145R_d.o objs/BUuFW_d.o objs/mTyBe_d.o  \
objs/qyxMa_d.o objs/tPNSe_d.o objs/SMWMn_d.o objs/CSfvL_d.o objs/NnYuc_d.o  \
objs/M6YDk_d.o objs/rv3PS_d.o objs/aZvMc_d.o objs/PbEsB_d.o objs/mTJir_d.o  \
objs/n9AgH_d.o objs/fNQuz_d.o objs/e5rai_d.o objs/grtRb_d.o objs/d318C_d.o  \
objs/A2xcs_d.o objs/DhfpD_d.o objs/ZN8HQ_d.o objs/nCEtB_d.o objs/Tbe3K_d.o  \
objs/arn13_d.o objs/kFNLQ_d.o objs/MYE8n_d.o objs/t6Qtc_d.o objs/NnNIf_d.o  \
objs/eyfz0_d.o objs/ypyt7_d.o objs/VHiQ4_d.o objs/BBL3M_d.o objs/LZ9y6_d.o  \
objs/nDJcM_d.o objs/VFW6S_d.o objs/AueWP_d.o objs/WVyuF_d.o objs/PPWFq_d.o  \
objs/r13jF_d.o objs/eWEwI_d.o objs/tuIB6_d.o objs/jQVBD_d.o objs/bThbz_d.o  \
objs/Gh0dk_d.o objs/hCpVP_d.o objs/AJr7r_d.o objs/qEAwn_d.o objs/GLPFf_d.o  \
objs/F95RV_d.o objs/nFcte_d.o objs/fqL7Y_d.o objs/NRkDY_d.o objs/M3t2y_d.o  \
objs/kMQT1_d.o objs/jYQpK_d.o objs/r2UZV_d.o objs/SJqfw_d.o objs/SbPJe_d.o  \
objs/usnnz_d.o objs/ehMdh_d.o objs/tkxhG_d.o objs/mbdx2_d.o objs/Gua3D_d.o  \
objs/K8xZ0_d.o objs/WyS0x_d.o objs/ay1Bn_d.o objs/Au7Vv_d.o objs/SgniT_d.o  \
objs/vEC6P_d.o objs/a6ari_d.o objs/y95ka_d.o objs/txAaE_d.o objs/jj2KY_d.o  \
objs/NPLgV_d.o objs/WtwyT_d.o objs/SpYgI_d.o objs/fPZ7f_d.o objs/Zp6Rk_d.o  \
objs/b4Qa5_d.o objs/HZfsa_d.o objs/uiBC0_d.o objs/jegsF_d.o objs/CkvBj_d.o  \
objs/EwT6I_d.o objs/w2sz1_d.o objs/YiDib_d.o objs/FfyV6_d.o objs/hxsxh_d.o  \
objs/fhL1z_d.o objs/HyGVH_d.o objs/c6HV9_d.o objs/grzAh_d.o objs/LkLxL_d.o  \
objs/zGpqe_d.o objs/vhqM5_d.o objs/u6DVK_d.o objs/ztFM4_d.o objs/CZhsi_d.o  \
objs/fviLL_d.o objs/Cxz3S_d.o objs/xQ3EW_d.o objs/Hsxtq_d.o objs/J6d3Q_d.o  \
objs/BGSKj_d.o objs/NpqSW_d.o objs/Du34z_d.o objs/PBsc7_d.o objs/RsMdD_d.o  \
objs/yswdU_d.o objs/s80jM_d.o objs/PtubC_d.o objs/jfcDP_d.o objs/x88Vd_d.o  \
amcQwB.o objs/kzijH_d.o objs/RaLuI_d.o objs/DMu5y_d.o objs/gsRQh_d.o  \
objs/hIkrk_d.o objs/Fusq8_d.o objs/T6IIT_d.o objs/ImS2n_d.o objs/jPcY5_d.o  \
objs/nG0Gi_d.o objs/BjLbH_d.o objs/fHm4i_d.o objs/IGStm_d.o objs/qaF8U_d.o  \
objs/SSxyL_d.o objs/fQfhs_d.o objs/HiDyg_d.o objs/vwaBf_d.o objs/zEZsG_d.o  \
objs/vjfuM_d.o objs/kg7Y3_d.o objs/c3AZi_d.o objs/aue1b_d.o objs/Q9Ygk_d.o  \
objs/UKmIj_d.o objs/q0DLq_d.o objs/reYIK_d.o objs/Mssj6_d.o objs/CauIB_d.o  \
objs/HUWGU_d.o objs/z7LiQ_d.o objs/uvqgb_d.o objs/CRk1e_d.o objs/tF3xC_d.o  \
objs/bzeja_d.o objs/HkiH0_d.o objs/KAxW4_d.o objs/cNpVh_d.o objs/suRL6_d.o  \
objs/pHNQR_d.o objs/UTFqT_d.o objs/tNvGj_d.o objs/QRxPR_d.o objs/vDnM2_d.o  \
objs/j1daB_d.o objs/BPGKY_d.o objs/ZDza6_d.o objs/RhZD1_d.o objs/n88gv_d.o  \
objs/FdeTV_d.o objs/HiUjT_d.o objs/GaZPH_d.o objs/t833Z_d.o objs/v0ksd_d.o  \
objs/jxDvJ_d.o objs/NnFqr_d.o objs/RY9dF_d.o objs/jSG5z_d.o objs/C1T3Y_d.o  \
objs/bMdmk_d.o objs/zvBT2_d.o objs/i2Z8j_d.o objs/cD9jV_d.o objs/SE20h_d.o  \
objs/E4Ayq_d.o objs/CIYxC_d.o objs/TfDgs_d.o objs/xbtnQ_d.o objs/Qps6P_d.o  \
objs/rxfJg_d.o objs/QF1Ae_d.o objs/TQ55I_d.o objs/p8ELQ_d.o objs/qbQhN_d.o  \
objs/ItbJp_d.o objs/yKjaI_d.o objs/RTyGs_d.o objs/R5a21_d.o objs/FvDN2_d.o  \
objs/cxfhu_d.o objs/Fc71k_d.o objs/dyZYk_d.o objs/nHgL7_d.o objs/FbZ6b_d.o  \
objs/F27uf_d.o objs/AaSnW_d.o objs/rwHfF_d.o objs/ak7Qh_d.o objs/Eketa_d.o  \
objs/gZC4Q_d.o objs/FP7vH_d.o objs/mjd8k_d.o objs/ff5sB_d.o objs/E8SEL_d.o  \
objs/x6HPU_d.o objs/yQjvZ_d.o objs/NTji4_d.o objs/D9Zhb_d.o objs/QFckQ_d.o  \
objs/AZ7hv_d.o objs/RsnLk_d.o objs/qysF6_d.o objs/QhgJ1_d.o objs/SZvDs_d.o  \
objs/zdinH_d.o objs/WdJ0P_d.o objs/KjMt5_d.o objs/rLGwW_d.o objs/Sa7Uv_d.o  \
objs/ZqQi2_d.o objs/YUfqj_d.o objs/T3jG5_d.o objs/zBJg9_d.o objs/YsuQY_d.o  \
objs/k8f1L_d.o objs/SxEDS_d.o objs/tfyYk_d.o objs/Ne1xV_d.o objs/qqLtz_d.o  \
objs/FABmH_d.o objs/SEFhv_d.o objs/AxjNQ_d.o objs/nFCgP_d.o objs/sM4pd_d.o  \
objs/c2YpQ_d.o objs/rQzvu_d.o objs/Ak6pb_d.o objs/psgd8_d.o objs/bASxv_d.o  \
objs/we7kj_d.o objs/R3a1P_d.o objs/juj8F_d.o objs/R25uE_d.o objs/PiF1N_d.o  \
objs/wWqej_d.o objs/c4ApD_d.o objs/uGjy8_d.o objs/stRp5_d.o objs/LiVhg_d.o  \
objs/tts03_d.o objs/Nt7Tj_d.o objs/gGaIt_d.o objs/u72bg_d.o objs/cEcNN_d.o  \
objs/esRfY_d.o objs/tySCV_d.o objs/a9Mgy_d.o objs/D2gH3_d.o objs/aQ0UG_d.o  \
objs/YKMrw_d.o objs/fU36M_d.o objs/LLBBF_d.o objs/fsipy_d.o objs/ms2uh_d.o  \
objs/EwDev_d.o objs/dD6Fe_d.o objs/TAc7m_d.o objs/qNyTq_d.o objs/pyU2U_d.o  \
objs/qVbvA_d.o objs/qupMR_d.o objs/CH4rU_d.o objs/SVdUx_d.o objs/k9ah3_d.o  \
objs/S9AKx_d.o objs/zctCK_d.o objs/zIQFd_d.o objs/WgQmC_d.o objs/ZhnV1_d.o  \
objs/Z9FG2_d.o objs/NzQWF_d.o objs/rqamY_d.o objs/SM5Bf_d.o objs/ppfiy_d.o  \
objs/JPdJD_d.o objs/s9N4H_d.o objs/ZDxRF_d.o objs/Z9Fb4_d.o objs/bsayK_d.o  \
objs/kLanD_d.o objs/SGVL9_d.o objs/vS8ph_d.o objs/Wz275_d.o objs/x3vKW_d.o  \
objs/kUvbM_d.o objs/a9CVV_d.o objs/FJi8F_d.o objs/nince_d.o objs/TZeau_d.o  \
objs/hRRRK_d.o objs/gEZJT_d.o objs/b2zMA_d.o objs/RyiqR_d.o objs/Gcdk1_d.o  \
objs/MDMK1_d.o objs/ygvhd_d.o objs/C0cuu_d.o objs/vGk6q_d.o objs/j8zYf_d.o  \
objs/rk4z8_d.o objs/GLz5K_d.o objs/We3Zu_d.o objs/QLQjj_d.o objs/WWJAL_d.o  \
objs/cCQrF_d.o objs/E2IdA_d.o objs/TH93q_d.o objs/Vg01W_d.o objs/Z4Dce_d.o  \
objs/aq3nc_d.o objs/cVKvW_d.o objs/BmsGu_d.o objs/j2qTn_d.o objs/Rdnh9_d.o  \
objs/vHuZn_d.o objs/MvUAz_d.o objs/yg8g0_d.o objs/Dvcje_d.o objs/uB7dU_d.o  \
objs/fWKaC_d.o objs/eGRgq_d.o objs/s7eQN_d.o objs/Sx3pG_d.o objs/BGWKV_d.o  \
objs/INFz8_d.o objs/m6nyB_d.o objs/SxYm5_d.o objs/E9BtG_d.o objs/QfjFn_d.o  \
objs/qRmnW_d.o objs/SK9tN_d.o objs/EDfdI_d.o objs/qK6Cv_d.o objs/LTMnF_d.o  \
objs/ACig5_d.o objs/xiGfH_d.o objs/fVU0u_d.o objs/H99ve_d.o objs/BM37A_d.o  \
objs/cf3am_d.o objs/grGJe_d.o objs/GHyCI_d.o objs/m22eg_d.o objs/uKuSu_d.o  \
objs/xCzMT_d.o objs/ZjgqY_d.o objs/sL9xS_d.o objs/VwjNj_d.o objs/Zvq8I_d.o  \
objs/mhyCk_d.o objs/YR8nj_d.o objs/sBF2i_d.o objs/YiNrx_d.o objs/Q4v2e_d.o  \
objs/Lt6bf_d.o objs/rDVuN_d.o objs/KjsDw_d.o objs/Rb8ph_d.o objs/zGMV0_d.o  \
objs/IJ1r9_d.o objs/Tqaqh_d.o objs/NsVPB_d.o objs/xAAyz_d.o objs/vJLqq_d.o  \
objs/urt1C_d.o objs/yp6pK_d.o objs/Dq3tD_d.o objs/Qkwre_d.o objs/Pvjrn_d.o  \
objs/p6yqK_d.o objs/uBivs_d.o objs/Qa0TD_d.o objs/Z0SSN_d.o objs/fr3n9_d.o  \
objs/uzsDM_d.o objs/YB51I_d.o objs/u2pha_d.o objs/tFbnG_d.o objs/FHFx0_d.o  \
objs/hpkx4_d.o objs/vxRzH_d.o objs/d5SdZ_d.o objs/qMtgk_d.o objs/SG9fF_d.o  \
objs/ksnS4_d.o objs/y77PZ_d.o objs/TJVyC_d.o objs/YCHNQ_d.o objs/Cahk1_d.o  \
objs/aHnTA_d.o objs/Hf6eL_d.o objs/JUF8r_d.o objs/FfhuY_d.o objs/n9n3i_d.o  \
objs/dnjbW_d.o objs/yhejB_d.o objs/AbFKM_d.o objs/Emc1h_d.o objs/mk0NH_d.o  \

VCS_ARC1 =_csrc1.so

VCS_OBJS1 =objs/irUFW_d.o objs/R6NGG_d.o objs/Zqhgr_d.o objs/TTRCa_d.o objs/chpSI_d.o  \
objs/ihGk8_d.o objs/Ra1ES_d.o objs/iZ827_d.o objs/FIIah_d.o objs/r8IWB_d.o  \
objs/gBkCy_d.o objs/NA3wJ_d.o objs/ap6vz_d.o objs/WGyjd_d.o objs/VPyVy_d.o  \
objs/vUS1C_d.o objs/tJtKK_d.o objs/YeJE3_d.o objs/rdk6s_d.o objs/Ti0bc_d.o  \
objs/yY5km_d.o objs/L5tgU_d.o objs/TAbzw_d.o objs/Z7pm0_d.o objs/KTRw7_d.o  \
objs/fp8jY_d.o objs/xLBVz_d.o objs/U6nmz_d.o objs/VCiBQ_d.o objs/w2L9P_d.o  \
objs/Hg4FG_d.o objs/HnqpA_d.o objs/ba8qx_d.o objs/MxjaV_d.o objs/HQeZw_d.o  \
objs/mqktR_d.o objs/japx2_d.o objs/CcrTq_d.o objs/HYJge_d.o objs/p0h27_d.o  \
objs/QRsac_d.o objs/Egh5n_d.o objs/vyFjE_d.o objs/DuhRY_d.o objs/bN2N0_d.o  \
objs/yZkyQ_d.o objs/StW6r_d.o objs/LBMnI_d.o 



%.o: %.c
	$(CC_CG) $(CFLAGS_CG) -c -o $@ $<

$(VCS_ARC0) : $(VCS_OBJS0)
	$(PIC_LD) -m elf_i386 -shared  -o .//../simv.daidir//$(VCS_ARC0) $(VCS_OBJS0)
	rm -f $(VCS_ARC0)
	@ln -sf .//../simv.daidir//$(VCS_ARC0) $(VCS_ARC0)


$(VCS_ARC1) : $(VCS_OBJS1)
	$(PIC_LD) -m elf_i386 -shared  -o .//../simv.daidir//$(VCS_ARC1) $(VCS_OBJS1)
	rm -f $(VCS_ARC1)
	@ln -sf .//../simv.daidir//$(VCS_ARC1) $(VCS_ARC1)

CU_UDP_OBJS = \
objs/udps/Ux6u8.o objs/udps/ZycYe.o objs/udps/Z0GwU.o objs/udps/cfY91.o objs/udps/pNeY8.o  \
objs/udps/Hz6M0.o objs/udps/YffJQ.o objs/udps/E3u6e.o objs/udps/mSfbM.o objs/udps/c8ZbZ.o  \
objs/udps/u96GI.o objs/udps/JzzcM.o objs/udps/KQQ1H.o objs/udps/Aw1WF.o objs/udps/Z0bgF.o  \
objs/udps/mTxzr.o objs/udps/y4a7N.o objs/udps/BcyNP.o objs/udps/CZfJ6.o objs/udps/cdUtU.o  \
objs/udps/e37Qb.o objs/udps/LSJP2.o objs/udps/K0FAD.o objs/udps/Gjg4Z.o objs/udps/hg6Rm.o  \
objs/udps/JgPNT.o 

CU_LVL_OBJS = \
SIM_l.o 

MAIN_OBJS = \
objs/amcQw_d.o 

CU_OBJS = $(MAIN_OBJS) $(ARCHIVE_OBJS) $(VCS_ARC0) $(VCS_ARC1) $(CU_UDP_OBJS) $(CU_LVL_OBJS)

